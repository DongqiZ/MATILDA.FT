#include "hip/hip_runtime.h"
// Copyright (c) 2023 University of Pennsylvania
// Part of MATILDA.FT, released under the GNU Public License version 2 (GPLv2).


#include "Extraforce_lewis_serial.h"
#include "Extraforce_dynamic.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include "globals.h"
#include <thrust/copy.h>
#include <thrust/shuffle.h>
#include <thrust/random.h>
#include <cmath>
#include <random>
#include <stdio.h>
#include <ctime>


#define EPSILON 1.0e-10

using namespace std;


LewisSerial::~LewisSerial() { return; }

LewisSerial::LewisSerial(istringstream &iss) : ExtraForce(iss)
{

    DynamicBonds.push_back(this);


    readRequiredParameter(iss, nlist_name);
	nlist_index = get_nlist_id(nlist_name);
    nlist = dynamic_cast<NListBonding*>(NLists.at(nlist_index));

    cout << "LewisSerial bonds active!" << endl;
    cout <<"N-list index: " << nlist_index << ", n-list name: " << nlist_name << endl;


    readRequiredParameter(iss, k_spring);
    cout << "k_spring: " << k_spring << endl;
    readRequiredParameter(iss, e_bond);
    cout << "e_bond: " << e_bond << endl;
    readRequiredParameter(iss, r0);
    cout << "r0: " << r0 << endl;
    readRequiredParameter(iss, qind);
     cout << "qind: " << qind << endl;
    readRequiredParameter(iss, bond_freq);
    cout << "bond_freq: " << bond_freq << endl;
    readRequiredParameter(iss, bond_log_freq);
    cout << "bond_log_freq: " << bond_log_freq << endl;
    readRequiredParameter(iss, file_name);
    cout << "output_file: " << file_name << endl;
    readRequiredParameter(iss, offset);
    cout << "offset: " << offset << endl;

    iss >> ramp_string;
    if (ramp_string == "ramp"){
        std::cout << "Energy ramp activated!" << std::endl;

        iss >> e_bond_final;
        iss >> ramp_reps;
        iss >> ramp_t_end;

        ramp_interval = ceil(float(ramp_t_end)/float(ramp_reps));
        ramp_counter = 0;
        RAMP_FLAG = 1;
        delta_e_bond = (e_bond_final - e_bond)/float(ramp_reps);
        std::cout << "Final energy: "<< e_bond_final <<", in " << ramp_reps <<" intervals of " << ramp_interval << " time steps" << std::endl;
    }
    else{
        RAMP_FLAG = 0;
    }

    cout << "Group size: " << group->nsites << endl;
    cout << "Donors: " << nlist->n_donors << endl;
    cout << "Acceptors: " << nlist->n_acceptors << endl;



    d_BONDS.resize(2 * group->nsites); 
    BONDS.resize(2 * group->nsites);     

    // BONDS - stores bonding information
    // group_id [n_bonds] [bonded_parter_group_id]

    for (int j = 0; j < group->nsites; j++)
    {
        BONDS[2 * j] = 0;
        BONDS[2 * j + 1] = -1;
    }

    d_BONDS = BONDS;

    d_VirArr.resize(5 * group->nsites); 
    for (int i = 0; i < 5 * group->nsites; ++i){
        d_VirArr[i] = 0.0f;
    }

    d_BONDED.resize( group->nsites);
    d_FREE.resize( group->nsites);

    BONDED.resize( group->nsites);
    FREE.resize( group->nsites);


    for (int i = 0; i < nlist->n_donors; ++i){
        FREE[i] = nlist->d_DONORS[i];
        BONDED[i] = -1;
    }

    n_bonded = 0;
    n_free = nlist->n_donors;

    d_FREE = FREE;
    d_BONDED = BONDED;

    GRID = ceil(((float)nlist->n_donors)/float(threads));
}


void LewisSerial::AddExtraForce()
{   

    if (RAMP_FLAG == 1 && ramp_counter < ramp_reps && step % ramp_interval == 0 && step > 0){
        e_bond += delta_e_bond;
        std::cout << "At step: " << step <<" increased e_bond to: " << e_bond << std::endl;
        ++ramp_counter;
    }



    if (step % bond_freq == 0 && step >= bond_freq){
        for (int loops = 0; loops < int(bond_freq/10); ++loops){
            int rnd = random()%2; //decide move sequence

            if (rnd == 0){
                if (n_free > 0){

                    prepareDensityFields();
                    MasterCharge->CalcCharges();
                    MasterCharge->CalcEnergy();

                    int rndid = random()%(n_free);
                    d_make_bonds_lewis_serial<<<1, 1>>>(d_x,d_f,
                        d_BONDS.data(),
                        nlist->d_RN_ARRAY.data(), nlist->d_RN_ARRAY_COUNTER.data(),
                        d_FREE.data(), d_VirArr.data(), n_free,
                        nlist->nncells, nlist->ad_hoc_density,
                        group->d_index.data(), group->nsites, d_states,
                        k_spring, e_bond, r0, nlist->r_n, qind, d_L, d_Lh, Dim,d_charges,
                        grid_per_partic,d_electrostatic_potential,d_grid_inds,d_grid_W, rndid);

                    hipMemcpy(charges, d_charges, ns * sizeof(float), hipMemcpyDeviceToHost);
                    check_cudaError("Make bonds lewis");

                    n_bonded = 0;
                    n_free = 0;

                    BONDS = d_BONDS;
                    for (int i = 0; i < group->nsites; ++i){
                        if (nlist->AD[i] == 1 && BONDS[2*i] == 1){
                            BONDED[n_bonded++] = i;
                        }
                        else if (nlist->AD[i] == 1 && BONDS[2*i] == 0){
                            FREE[n_free++] = i;
                        }
                    }


                    d_BONDED = BONDED;
                    d_FREE = FREE;

                    check_cudaError("Update charges lewis bonds");

                } 


                
                if (n_bonded > 0){

                    int rndid = random()%(n_bonded);

                    prepareDensityFields();
                    MasterCharge->CalcCharges();
                    MasterCharge->CalcEnergy();
                    d_break_bonds_lewis_serial<<<1,1>>>(d_x,
                        d_BONDS.data(),
                        nlist->d_RN_ARRAY.data(), nlist->d_RN_ARRAY_COUNTER.data(),
                        d_BONDED.data(),n_bonded,
                        nlist->nncells, nlist->ad_hoc_density,
                        group->d_index.data(), group->nsites, d_states,
                        k_spring, e_bond, r0, qind, d_L, d_Lh, Dim, d_charges,
                        grid_per_partic,d_electrostatic_potential,d_grid_inds,d_grid_W, rndid);

                hipMemcpy(charges, d_charges, ns * sizeof(float), hipMemcpyDeviceToHost);

                    n_bonded = 0;
                    n_free = 0;

                    BONDS = d_BONDS;
                    for (int i = 0; i < group->nsites; ++i){
                        if (nlist->AD[i] == 1 && BONDS[2*i] == 1){
                            BONDED[n_bonded++] = i;
                        }
                        else if (nlist->AD[i] == 1 && BONDS[2*i] == 0){
                            FREE[n_free++] = i;
                        }
                    }

                    d_BONDED = BONDED;
                    d_FREE = FREE;


                    //Update charges

                    check_cudaError("Break bonds lewis");
                } 
            }

            else {
                if (n_bonded > 0){

                    prepareDensityFields();
                    MasterCharge->CalcCharges();
                    MasterCharge->CalcEnergy();

                    int rndid = random()%(n_bonded);
                    d_break_bonds_lewis_serial<<<1,1>>>(d_x,
                        d_BONDS.data(),
                        nlist->d_RN_ARRAY.data(), nlist->d_RN_ARRAY_COUNTER.data(),
                        d_BONDED.data(),n_bonded,
                        nlist->nncells, nlist->ad_hoc_density,
                        group->d_index.data(), group->nsites, d_states,
                        k_spring, e_bond, r0, qind, d_L, d_Lh, Dim, d_charges,
                        grid_per_partic,d_electrostatic_potential,d_grid_inds,d_grid_W, rndid);

                hipMemcpy(charges, d_charges, ns * sizeof(float), hipMemcpyDeviceToHost);


                    n_bonded = 0;
                    n_free = 0;

                    BONDS = d_BONDS;
                    for (int i = 0; i < group->nsites; ++i){
                        if (nlist->AD[i] == 1 && BONDS[2*i] == 1){
                            BONDED[n_bonded++] = i;
                        }
                        else if (nlist->AD[i] == 1 && BONDS[2*i] == 0){
                            FREE[n_free++] = i;
                        }
                    }

                    d_BONDED = BONDED;
                    d_FREE = FREE;

                    //Update charges
                    check_cudaError("Break bonds lewis");

                } // if n_free > 0


                if (n_free > 0){

                    int rndid = random()%(n_free);

                    prepareDensityFields();
                    MasterCharge->CalcCharges();
                    MasterCharge->CalcEnergy();
                    d_make_bonds_lewis_serial<<<1, 1>>>(d_x,d_f,
                        d_BONDS.data(),
                        nlist->d_RN_ARRAY.data(), nlist->d_RN_ARRAY_COUNTER.data(),
                        d_FREE.data(), d_VirArr.data(), n_free,
                        nlist->nncells, nlist->ad_hoc_density,
                        group->d_index.data(), group->nsites, d_states,
                        k_spring, e_bond, r0, nlist->r_n, qind, d_L, d_Lh, Dim, d_charges,
                        grid_per_partic,d_electrostatic_potential,d_grid_inds,d_grid_W,rndid);

                hipMemcpy(charges, d_charges, ns * sizeof(float), hipMemcpyDeviceToHost);

                    // update the bonded array

                    n_bonded = 0;
                    n_free = 0;

                    BONDS = d_BONDS;
                    for (int i = 0; i < group->nsites; ++i){
                        if (nlist->AD[i] == 1 && BONDS[2*i] == 1){
                            BONDED[n_bonded++] = i;
                        }
                        else if (nlist->AD[i] == 1 && BONDS[2*i] == 0){
                            FREE[n_free++] = i;
                        }
                    }

                    d_BONDED = BONDED;
                    d_FREE = FREE;

                    //Update charges

                    check_cudaError("Make bonds lewis");

                }
            }
            prepareDensityFields();
            MasterCharge->CalcCharges();
            MasterCharge->CalcEnergy();
        }
    } // end if (step % lewis_bond_freq == 0 && step >= bond_freq)

    if(step >= bond_freq){
        d_update_forces<<<GRID, threads>>>(d_x, d_f, d_L, d_Lh,
            k_spring, e_bond, r0,
            d_BONDS.data(), d_BONDED.data(), d_VirArr.data(), n_bonded,
            group->d_index.data(), group->nsites, Dim);
    }

    if (step == 0){
        const char* fname = file_name.c_str();
        remove(fname);
    }

    if (step % bond_log_freq == 0 && step >= offset)
    {
        LewisSerial::WriteBonds();
    }
}


/*
Updates forces acting on particles due to LewisSerial bonds
*/



__global__ void d_make_bonds_lewis_serial(
    const float *x,
    float* f,
    thrust::device_ptr<int> d_BONDS,
    thrust::device_ptr<int> d_RN_ARRAY,
    thrust::device_ptr<int> d_RN_ARRAY_COUNTER,
    thrust::device_ptr<int> d_FREE,
    thrust::device_ptr<float> d_VirArr,
    int n_free,
    int nncells,
    int ad_hoc_density,
    thrust::device_ptr<int> d_index, 
    const int ns,        
    hiprandState *d_states,
    float k_spring,
    float e_bond,
    float r0,
    float r_n,
    float qind,
    float *L,
    float *Lh,
    int D,
    float* d_charges,
    int grid_per_partic,
    float* d_electrostatic_potential,
    int* d_grid_inds,
    float* d_grid_W,
    int rndid)

{

    int tmp_ind = rndid;



    int list_ind = d_FREE[tmp_ind];
    int ind = d_index[list_ind];

    hiprandState l_state;
    l_state = d_states[ind];
    d_states[ind] = l_state;

    int lnid;
    int c = d_RN_ARRAY_COUNTER[list_ind];

    if (c != 0){
        l_state = d_states[ind];
        int r = (int)((hiprand_uniform(&l_state) * (INT_MAX + .999999)));
        d_states[ind] = l_state;
        lnid = d_RN_ARRAY[list_ind * ad_hoc_density * nncells + r%c];
    }
    else{
        return;
        }

    if (atomicCAS(&d_BONDS.get()[lnid * 2], 0, -1) == 0){ //lock the particle to bond with

        double dr_sq = 0.0;
        double dr0 = 0.0;
        double dr_arr[3];
        double delr = 0.0;
        double dU = 0.0;


        int nid = d_index[lnid];

        hiprandState l_state;
        l_state = d_states[ind];
        float rnd = hiprand_uniform(&l_state);
        d_states[ind] = l_state;

        for (int j = 0; j < D; j++){

            dr0 = x[ind * D + j] - x[nid * D + j];
            if (dr0 >  Lh[j]){dr_arr[j] = -1.0 * (L[j] - dr0);}
            else if (dr0 < -1.0 * Lh[j]){dr_arr[j] = (L[j] + dr0);}
            else{dr_arr[j] = dr0;}
            dr_sq += dr_arr[j] * dr_arr[j];
        }

        double mdr = sqrt(dr_sq); //distance
        if (mdr > EPSILON){ 
            delr = mdr - r0; //distance - r_eq
            dU = delr * delr * k_spring;
        }
        else
        {
            dU = 0.0;
            mdr = 0.0;
        }


        /*
            Extra acceptance criterion due to electrostatics
        */
    

       for (int grid_ct=0; grid_ct < grid_per_partic; ++grid_ct){

            dU += qind * d_electrostatic_potential[d_grid_inds[ind * grid_per_partic + grid_ct]] * d_grid_W[ind * grid_per_partic + grid_ct];

            dU -= qind * d_electrostatic_potential[d_grid_inds[nid * grid_per_partic + grid_ct]] * d_grid_W[nid * grid_per_partic + grid_ct];

       }



        if (mdr <= r_n && rnd < exp(-dU + e_bond))
        {
            atomicExch(&d_BONDS.get()[list_ind * 2], 1);
            atomicExch(&d_BONDS.get()[lnid * 2], 1);

            atomicExch(&d_BONDS.get()[list_ind * 2 + 1], lnid);
            atomicExch(&d_BONDS.get()[lnid * 2 + 1], list_ind);

            // printf("Ind Nid ch1 ch2: %d %d %f %f\n", ind,nid, d_charges[ind],d_charges[nid]);    
            d_charges[ind] += qind; 
            d_charges[nid] -= qind;
            // printf("Ind Nid ch1 ch2: %d %d %f %f\n", ind,nid, d_charges[ind],d_charges[nid]);   

        }

        else
        {

            atomicExch(&d_BONDS.get()[list_ind * 2], 0);
            atomicExch(&d_BONDS.get()[lnid * 2], 0);

            atomicExch(&d_BONDS.get()[list_ind * 2 + 1], -1);
            atomicExch(&d_BONDS.get()[lnid * 2 + 1], -1);

        }
    } // if particle got locked
}






__global__ void d_break_bonds_lewis_serial(
    const float *x,
    thrust::device_ptr<int> d_BONDS,
    thrust::device_ptr<int> d_RN_ARRAY,
    thrust::device_ptr<int> d_RN_ARRAY_COUNTER,
    thrust::device_ptr<int> d_BONDED,
    int n_bonded,
    int nncells,
    int ad_hoc_density,
    thrust::device_ptr<int> d_index, 
    const int ns,        
    hiprandState *d_states,
    float k_spring,
    float e_bond,
    float r0,
    float qind,
    float *L,
    float *Lh,
    int D,
    float* d_charges,
    int grid_per_partic,
    float* d_electrostatic_potential,
    int* d_grid_inds,
    float* d_grid_W,
    int rndid)


{
    int tmp_ind = rndid;


    int list_ind = d_BONDED[tmp_ind];
    int ind = d_index[list_ind];


    hiprandState l_state;
    l_state = d_states[ind];
    float rnd = hiprand_uniform(&l_state);
    d_states[ind] = l_state;

    int lnid = d_BONDS[list_ind * 2 + 1];
    int nid = d_index[lnid];

    double dr_sq = 0.0;
    double dr0;
    double dr_arr[3];
    double delr;
    double dU;



    for (int j = 0; j < D; j++){

        dr0 = x[ind * D + j] - x[nid * D + j];
        if (dr0 >  Lh[j]){dr_arr[j] = -1.0 * (L[j] - dr0);}
        else if (dr0 < -1.0 * Lh[j]){dr_arr[j] = (L[j] + dr0);}
        else{dr_arr[j] = dr0;}
        dr_sq += dr_arr[j] * dr_arr[j];
    }

    double mdr = sqrt(dr_sq); //distance
    if (mdr > EPSILON){ 
        delr = mdr - r0; //distance - r_eq
        dU = delr * delr * k_spring;
    }
    else
    {
        dU = 0.0;
    }
    // printf("Ubond1_:%d %f\n",ind, dU);
    for (int grid_ct=0; grid_ct < grid_per_partic; ++grid_ct){

        dU -= qind * d_electrostatic_potential[d_grid_inds[ind * grid_per_partic + grid_ct]] * d_grid_W[ind * grid_per_partic + grid_ct];

        dU += qind * d_electrostatic_potential[d_grid_inds[nid * grid_per_partic + grid_ct]] * d_grid_W[nid * grid_per_partic + grid_ct];


    }

    // printf("Ubond2_:%d %f\n",ind, dU);

    if (rnd <= exp(dU - e_bond))
    {
        atomicExch(&d_BONDS.get()[list_ind * 2], 0);
        atomicExch(&d_BONDS.get()[lnid * 2], 0);

        atomicExch(&d_BONDS.get()[list_ind * 2 + 1], -1);
        atomicExch(&d_BONDS.get()[lnid * 2 + 1], -1);

        // printf("Ind Nid ch1 ch2: %d %d %f %f\n", ind,nid, d_charges[ind],d_charges[nid]);    
        d_charges[ind] -= qind; 
        d_charges[nid] += qind;
        // printf("Ind Nid ch1 ch2: %d %d %f %f\n", ind,nid, d_charges[ind],d_charges[nid]);   


    }
}

void LewisSerial::WriteBonds(void)
{

    this->BONDS = d_BONDS;
    ofstream bond_file;
    // bond_file.open(file_name, ios::out | ios::app);


    // bond_file << "TIMESTEP: " << global_step << " " << n_bonded << " " << n_free << " " << n_free + n_bonded << endl;
    // for (int j = 0; j < group->nsites; ++j)
    // {
    //     if (BONDS[2 * j + 1] != -1 && nlist->AD[j] == 1)
    //     {
    //         bond_file << group->index[j] + 1 << " " << this->group->index[BONDS[2 * j + 1]] + 1 << endl;
    //     }
    // }
    // bond_file.close();
    bond_file.open("bond_data", ios::out | ios::app);

    bond_file << global_step << " " << float(n_bonded)/float(n_free + n_bonded) << " " << MasterCharge->energy << endl;
    bond_file.close();
}


void LewisSerial::UpdateVirial(void){

    VirArr = d_VirArr;

    // VirArr [5 * group size]
    // d_BONDED: stores group indices (not global index) of the bonded DONOR particles
    // Columns 0-2 displacement vector from the DONOR to the ACCEPTOR
    // Column 3 stores the current  bond energy
    // Column 4 stores mf
    // d_VirArr is updated in UpdateForces or make_bonds routines

    for (int k = 0; k < n_bonded; ++k){
        int j = d_BONDED[k];
        Udynamicbond += VirArr[j*5+3];
        float mf = VirArr[j*5+4];

        bondVir[0] += -mf * VirArr[j*5] * VirArr[j*5];
        bondVir[1] += -mf * VirArr[j*5+1] * VirArr[j*5+1];
        if ( Dim == 2 )
            bondVir[2] += -mf * VirArr[j*5] * VirArr[j*5+1];
        else if (Dim == 3)
        {
            bondVir[2] += -mf * VirArr[j*5+2] * VirArr[j*5+2];
            bondVir[3] += -mf * VirArr[j*5] * VirArr[j*5+1];
            bondVir[4] += -mf * VirArr[j*5] * VirArr[j*5+2];
            bondVir[5] += -mf * VirArr[j*5+1] * VirArr[j*5+2];
        }
    }
}
