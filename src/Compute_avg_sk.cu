// Copyright (c) 2023 University of Pennsylvania
// Part of MATILDA.FT, released under the GNU Public License version 2 (GPLv2).


#include "Compute_avg_sk.h"
#include "globals.h"
#include <algorithm>
#include "global_templated_functions.h"

using namespace std;

Avg_sk::~Avg_sk(){}

Avg_sk::Avg_sk(istringstream& iss) : Compute(iss)
{
    style = "avg_s_k";
    readRequiredParameter(iss, particle_type);

    num_data_pts = 0;

    set_optional_args(iss);

    cout << "  Calculating <S(k)> for component " << particle_type + 1 << " every " << this->compute_freq << " steps after " << this->compute_wait << " steps have passed." << endl;
}

// Generate AllocStorage for the Avg_sk class

void Avg_sk::allocStorage()
{
    this->cpx.resize(M);
    fill(this->cpx.begin(), this->cpx.end(), 0.0f);

    cout << " this->cpx has initial size " << this->cpx.size() << " and capacity " << this->cpx.capacity() << endl;

}

void Avg_sk::writeResults(){

  for (int i = 0; i < M; i++) {
    if ( num_data_pts > 0)
      k_tmp[i] = this->cpx[i] / float(num_data_pts);
    else
      k_tmp[i] = 0.0f;
  }

  char nm[50];

  // compute_id is used in the name instead of "type" in case multiple
  // computes operate on the same type
  sprintf(nm, "avg_sk_%d.dat", compute_id);
  write_kspace_data(nm, k_tmp);
}



void Avg_sk::doCompute(){

    // Extract the density of the relevant type
    d_prepareDensity<<<M_Grid, M_Block>>>(particle_type, d_all_rho, d_cpx1, M);
    check_cudaError("Compute->doCompute.style = avg_sk prepare density");

    // fourier from d_cpx1 to d_cpx2 forward
    hipfftExecC2C(fftplan, d_cpx1, d_cpx2, HIPFFT_FORWARD);
    check_cudaError("Compute->doCompute.style = avg_sk cufftExec");


    // Multiply by the complex conjugate and scale by 1/M
    // Store it in d_cpx1 as the values inside are not needed at this point
    d_multiplyComplex<<<M_Grid, M_Block>>> (d_cpx2, d_cpx2, d_cpx1, M);
    check_cudaError("Compute->doCompute.style = avg_sk multiplyComplex");


    // Copy data to host and store
    // NOTE: this should probably be stored on the device and only 
    // communicated when writing, but may be OK for now.
    hipMemcpy(cpx1, d_cpx1, M * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    for (int i = 0; i < M; i++)
        this->cpx.at(i) += cpx1[i].x + I * cpx1[i].y;


    num_data_pts++;
}