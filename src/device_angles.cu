#include "hip/hip_runtime.h"
// Copyright (c) 2023 University of Pennsylvania
// Part of MATILDA.FT, released under the GNU Public License version 2 (GPLv2).


#include "device_utils.cuh"

__global__ void d_angles(
    const float* x,                 // [ns*Dim] particle positions
    float* f,                       // [ns*Dim] particle forces
    const float* angle_k,           // [nangle_types] force constants
    const float* angle_theta_eq,    // [nangle_types] equilibrium angle
    const int* angle_style,         // [nangle_types] angle style flag (0=WLC, 1=Harmonic)
    const int* n_angles,            // [ns] Number of angles for each particle
    const int* angle_type,          // [ns * MAX_ANGLES] angle potential style
    const int* angle_first,         // [ns * MAX_ANGLES] first index of particles involved
    const int* angle_mid,           // [ns * MAX_ANGLES] second index 
    const int* angle_end,           // [ns * MAX_ANGLES] final index
    const float* L, const float* Lh,// [Dim]
    const int ns, const int MAX_ANGLES, const int Dim ) {

        const int ind = blockIdx.x * blockDim.x + threadIdx.x;
        if ( ind >= ns )
            return;

        float lforce[3];
        for ( int j=0; j<Dim ; j++ ) 
            lforce[j] = 0.0f;

        for ( int i=0 ; i < n_angles[ind] ; i++ ) {
            int aind = ind * MAX_ANGLES + i;        // stacked index to access ns*MAX_ANGLES arrays

            int atp = angle_type[aind];
            int ai = angle_first[aind];
            int aj = angle_mid[aind];
            int ak = angle_end[aind];

            float fi[3], fk[3];
            for ( int j=0 ; j<Dim ; j++ ) 
              fi[j] = fk[j] = 0.f;

            // Define vector connecting first to mid
            float rij[3];
            float mrij2 = 0.0f;
            for ( int j=0 ; j<Dim ; j++ ) {
                rij[j] = x[ai * Dim + j] - x[aj * Dim + j];
                if ( rij[j] > Lh[j] ) rij[j] -= L[j];
                else if ( rij[j] < -Lh[j] )  rij[j] += L[j];

                mrij2 += rij[j] * rij[j];
            }
            if ( mrij2 < 1.0E-4f)
                continue;
            float mrij = sqrtf(mrij2);

            // Vector connecting end to mid
            float rkj[3];
            float mrkj2 = 0.0f;
            for ( int j=0 ; j<Dim ; j++ ) {
                rkj[j] = x[ak * Dim + j] - x[aj * Dim + j];
                if ( rkj[j] > Lh[j] ) rkj[j] -= L[j];
                else if ( rkj[j] < -Lh[j] )  rkj[j] += L[j];

                mrkj2 += rkj[j] * rkj[j];
            }
            if ( mrkj2 < 1.0E-4f)
                continue;
            float mrkj = sqrtf(mrkj2);

            // Define angle between bonds
            float dot = 0.0f;
            for ( int k=0 ; k < Dim ; k++ )
                dot += rij[k] * rkj[k];
            float cos_th = dot / mrij / mrkj ;



            // WLC potential derivative and forces //
            if ( angle_style[atp] == 0 ) {
              // Derivative of cos(\theta_ijk) w.r.t. r_i 
              float DcosDri[3];
              for ( int j=0 ; j<Dim ; j++ )
                  DcosDri[j] = rkj[j] / mrij / mrkj - cos_th * rij[j] / mrij2;
              
              // Derivative of cos(\theta_ijk) w.r.t. r_k 
              float DcosDrk[3];
              for ( int j=0 ; j<Dim ; j++ )
                  DcosDrk[j] = rij[j] / mrkj / mrij - cos_th * rkj[j] / mrkj2;

              float kang = angle_k[atp];

              // Define forces on i, k
              // In the future, this will be angle-style dependent
              for ( int j=0 ; j<Dim ; j++ ) {
                  fi[j] = -kang * DcosDri[j];
                  fk[j] = -kang * DcosDrk[j];
              }
            }// angle_style[0] == 0 , WLC




            // Harmonic angle potential forces
            else if ( angle_style[atp] == 1 ) {
              if ( cos_th < -1.f || cos_th > 1.f )
                continue;

              float theta = acosf(cos_th);
              float denom = sqrtf(1.f - cos_th * cos_th);
              float iSinTheta = 0.f;
              if ( denom > 1.0E-4)
                iSinTheta = 1.0 / denom;

              float dtheta = theta - angle_theta_eq[atp];
              float du_mag = 2.f * angle_k[atp] * dtheta;

              for ( int j=0 ; j<Dim ; j++ ) {
                  fi[j] = du_mag * iSinTheta * ( rkj[j] / mrij / mrkj - cos_th * rij[j] / mrij2 );
                  fk[j] = du_mag * iSinTheta * ( rij[j] / mrij / mrkj - cos_th * rkj[j] / mrkj2 );
              }

            }// Harmonic angle potential forces





            // Accumulate the force based on which particle
            // has index "ind"
            for ( int j=0 ; j<Dim ; j++ ) {
                if ( ind == ai )
                    lforce[j] += fi[j];
                
                else if ( ind == aj ) 
                    lforce[j] -= ( fi[j] + fk[j] );

                else if ( ind == ak ) 
                    lforce[j] += fk[j];
            }

        }// i=0:n_angles[ind]
    

    // Finally accumulate the force on the 
    // global force array
    for ( int j=0 ; j<Dim ; j++ ) 
        f[ind*Dim+j] += lforce[j];
}
