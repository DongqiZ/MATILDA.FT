#include "hip/hip_runtime.h"
// Copyright (c) 2023 University of Pennsylvania
// Part of MATILDA.FT, released under the GNU Public License version 2 (GPLv2).


#define MAIN
#include <string.h>
#include "globals.h"
#include "timing.h"
#include "random.h"
#include <vector>
#include <fstream>
#include "git-version.h"
#include "Box.h"
#include <mpi.h>
#include <algorithm>
#include <random>



using namespace std;

void forces(void);
void update_potentials(void);
void calc_properties(int);
void initialize(void);
void write_lammps_traj(void);
void write_gsd_traj(void);
void cuda_collect_x(void);
void cuda_collect_f(void);
void cuda_collect_rho(void);
void write_binary(void);
void write_data_header(std::string);
void set_write_status(void);
void init_binary_output(void);
void write_struc_fac(void);
void write_grid_data(const char*, float*);
void write_kspace_data(const char*, complex<float>*);
void write_kspace_cudaComplex(const char*, hipfftComplex*);
__global__ void d_prepareDensity(int, float*, hipfftComplex*, int);
int print_timestep();
ofstream dout;
void unstack_like_device(int id, int* nn);
void run_computes();
void run_frame_printing();

void run_fts_sim(void);
void set_ft_config(void);



using namespace std;

Box* BoxFactory(istringstream&);

__global__ void cu_random_posits(float*, float*, int, int, hiprandState*);


__global__ void d_real2complex(float*, hipfftComplex*, int);
__global__ void d_complex2real(hipfftComplex*, float*, float*, int);
__global__ void d_make_step(hipfftComplex* , float*, float*, int*, int, int);
__global__ void d_multiplyComplex(hipfftComplex*, hipfftComplex*,
	hipfftComplex*, int);

int main(int argc, char** argv)
{


/////// Initialize MPI ////////

    MPI_Init(&argc, &argv);
    MPI_Comm communicator = MPI_COMM_WORLD;

    int size, rank;
    MPI_Comm_size(communicator, &size);
    MPI_Comm_rank(communicator, &rank);
	srank = std::to_string(rank);

    const char* nl_rank = getenv("OMPI_COMM_WORLD_LOCAL_RANK");
    int node_local_rank = atoi(nl_rank);

    int num_devices = 0;
    hipGetDeviceCount(&num_devices);

    int device_id = node_local_rank % num_devices;
    hipSetDevice(device_id);

	int my_device_id;
	hipGetDevice(&my_device_id);

	char processor_name[MPI_MAX_PROCESSOR_NAME];
    int name_len;
    MPI_Get_processor_name(processor_name, &name_len);

	int replica_exchange_flag = 0;
	std::vector<int> replica_id_vec(size);
	std::vector<float> replica_E_vec;
	std::iota (std::begin(replica_id_vec), std::end(replica_id_vec), 0);

	old_E_arr = (float*)calloc(size, sizeof(float));
	current_E_arr = (float*)calloc(size, sizeof(float));

	// auto rng = std::default_random_engine {};
	std::default_random_engine rng;
	std::uniform_int_distribution<int> pick_me_please(0,size);
	int msg_tag;


	if ( argc < 2 ) {
		std::cout << "ERROR: simulation style not specified!" << std::endl;
		std::cout << "Execute matilda.ft as either\nmatilda.ft -particle\nfor a particle-based simulation or"<< std::endl;
		std::cout << "matilda.ft -ft\nfor a field-theoretic simulation." << std::endl;
		die("Insufficient arguments");
	}

	printf("\n\n\t\t##### MPI INFO #####\nName: %s\nGlobal Rank: %2d of %2d, Local Rank: %2d, GPU: %2d (%2d) of %2d\n\n",
		processor_name,rank, size, node_local_rank, device_id, my_device_id, num_devices);


	// hipStreamCreateWithFlags(&stream1,hipStreamNonBlocking);
	
	// printf("Git Version hash: %s\n", MY_GIT_VERSION);
	main_t_in = int(time(0));
	init_t_in = main_t_in;
	std::vector<std::string> string_vec;
	std::cout << std::flush;

	for (int i = 0; i < argc; i++)
	{
		std::string arg = argv[i];
		string_vec.push_back(arg);
	}

	if(string_vec[1] == "-ft"){
		std::cout << "Set simulation style to: FT" << std::endl;
		set_ft_config();
		input_file = "input";
	    ifstream in2(input_file);

	    string word, line, rname;
		while (!in2.eof()) {
			getline(in2, line);

			// Blank or commented line
			if (line.length() == 0 || line.at(0) == '#')
				continue;

			istringstream iss(line);
			// Loop over words in line
			while (iss >> word) {
				if( word == "box" ) {
				box.push_back(BoxFactory(iss));
				box.back()->readInput(in2);
				field_sim = 1;
				particle_sim = 0;
				}
			}
		}
		run_fts_sim();
		return 0;
	}

	else if (string_vec[1] == "-particle"){
		std::cout << "Set simulation style to: TILD" << std::endl;

		for (int i = 2; i < string_vec.size(); ++i) {
			if (string_vec[i] == "-in") {
				input_file = string_vec[++i];
			}
			if (string_vec[i] == "-replica"){
				replica_exchange_flag = 1;
				replica_freq = std::stoi(string_vec[i+1]);
				replica_file = string_vec[i+2];
			}
		}

		if (replica_exchange_flag == 1){
			std::fstream file;
			std::string dummy_wrd;
			file.open(replica_file.c_str());

			while(file >> dummy_wrd) {
				replica_E_vec.push_back(std::stof(dummy_wrd));
				}
			file.close();

			std::cout << "Replica IDs: " << std::endl;
			for (auto& i : replica_id_vec)
				std::cout << ' ' << i;
			std::cout << endl;

			std::cout << "Replica bond energies: " << std::endl;
			for (auto& i : replica_E_vec)
				std::cout << ' ' << i;
			std::cout << endl;
			current_E = replica_E_vec[rank];
			std::cout <<"My energy: " << current_E << std::endl;
			for (int j = 0; j < replica_E_vec.size(); ++j){
				old_E_arr[j] = replica_E_vec[j];
				current_E_arr[j] = replica_E_vec[j];
			}			
		}



		initialize();

		set_write_status();


		// Write initial positions to lammpstrj file
		cuda_collect_x();
		
		forces();
		//hipDeviceSynchronize();

		cuda_collect_rho();
		cuda_collect_x();

		run_frame_printing();

		calc_properties(1);  // 1 indicates to calculate virial pressure

		if (grid_freq > 0) {
			print_t_in = int(time(0));
			//hipDeviceSynchronize();

			for (int i = 0; i < ntypes; i++) {
				char nm[30];
				sprintf(nm, "rho%d.dat", i);
				write_grid_data(nm, Components[i].rho);
			}
				
			print_t_out = int(time(0));
			print_tot_time += print_t_out - print_t_in;
		}

		int die_flag = 0;

		init_t_out = int(time(0));


		cout << "ENTERING MAIN LOOP!!" << endl;
		///////////////////////////////////////
		// BEGINNING OF MAIN SIMULATION LOOP //
		///////////////////////////////////////

		for (step = 1, global_step = global_step + 1; step <= max_steps; step++, global_step++) {
			if (equil  && step >= equil_steps) {
				dout.close();
				equil = false;
				set_write_status();
			}

			if (replica_exchange_flag == 1 && step%replica_freq == 0){
				if (rank == 0){
					std::shuffle(replica_id_vec.begin(), replica_id_vec.end(), rng);
					std::cout << "Step " << step << " | Replica IDs: " << std::endl;
					for (auto& i : replica_id_vec)
						std::cout << ' ' << i;
					std::cout << endl;

					for(int j = 0; j < size; j = j + 2){

						int rid = replica_id_vec[j];
						int n_rid = replica_id_vec[j+1];

						current_E_arr[n_rid] = old_E_arr[rid];
						current_E_arr[rid] = old_E_arr[n_rid];

						}
					std::cout << "Energies: " << std::endl;
					for(int j = 0; j < size; j++){
						old_E_arr[j] = current_E_arr[j];
						std::cout << current_E_arr[j] << " ";
						}
					std::cout << std::endl;
					
				} // if rank == 0

				MPI_Bcast(current_E_arr,size,MPI_FLOAT,0,communicator);
				MPI_Barrier(communicator);

				current_E = current_E_arr[rank];
				std::cout << current_E << std::endl;
			} // if replice_freq % step == 0



				// if (step%100 == 0 && step > 0){
				// 	if (rank == 0){
				// 		oldT = T;
				// 		MPI_Send(&oldT,1,MPI_INT,1, 0,communicator);
				// 		MPI_Recv(&T, 1, MPI_INT, 1, 0, communicator, MPI_STATUS_IGNORE);

				// 	}
				// 	else if(rank == 1){
				// 		oldT = T;
				// 		MPI_Recv(&T, 1, MPI_INT, 0, 0, communicator, MPI_STATUS_IGNORE);
				// 		MPI_Send(&oldT,1,MPI_INT,0, 0,communicator);
				// 	}
				// }


			for (auto Iter: Groups){
				Iter->CheckGroupMembers();
			}

			for (auto Iter: Integrators){
				Iter->Integrate_1();
			}

			check_cudaError("Integrator step 1");	
			
			if ( NLists.size() > 0 ) {
				nList_t_in = time(0);
				for (auto Iter: NLists)
					Iter->MakeNList();
				check_cudaError("Error in N-lists");
				nList_tot_time += time(0) - nList_t_in;
			}		

			forces();
			
			if ( ExtraForces.size() > 0 ) {
				extraForce_t_in = time(0);
				for (auto Iter: ExtraForces)
					Iter->AddExtraForce();
				check_cudaError("extraForces");
				extraForce_tot_time += time(0) - extraForce_t_in;
			}

			for (auto Iter: Integrators){
				Iter->Integrate_2();
			}

			check_cudaError("Integrator step 2");
			
			// Run computes
			run_computes();	

			// Write frames if applicable
			run_frame_printing();

			// Write to log file, write compute results
			if (step % log_freq == 0) {
			print_t_in = int(time(0));
			//hipDeviceSynchronize();

			calc_properties(1);

			die_flag = print_timestep();

			for (auto Iter : Computes) {
				if (step > Iter->compute_wait) {
				Iter->writeResults();
				}
			}

			print_t_out = int(time(0));
			print_tot_time += print_t_out - print_t_in;
			if (die_flag) {
				break;
			}
			}

					// Finalize time step //
			update_potentials();

		}// main loop over steps



		// Write resume frame and finish //
		if (max_steps % log_freq != 0) {
			cuda_collect_x();
			write_lammps_traj();
			write_gsd_traj();
		}
	}// if -particle

    else {
        die("Invalid simulation style, argument must be '-ft' or '-particle'\n");
    }

	main_t_out = int(time(0));
	int dt = main_t_out - main_t_in;
	cout << "Total run time: " << dt / 60 << "m" << dt % 60 << "sec" << endl;
	
	dt = init_t_out - init_t_in;
	cout << "Total init time: " << dt / 60 << "m" << dt % 60 << "sec" << endl;
	
	dt = bond_tot_time;
	cout << "Bond E, P props on host: " << dt / 60 << "m" << dt % 60 << "sec" << endl;

	dt = print_tot_time;
	cout << "I/O + Comm time: " << dt / 60 << "m" << dt % 60 << "sec" << endl;

	dt = compute_tot_time;
	cout << "Computes time: " << dt / 60 << "m" << dt % 60 << "sec" << endl;

    dt = extraForce_tot_time;
	cout << "ExtraForces time: " << dt / 60 << "m" << dt % 60 << "sec" << endl;

    if ( DPD_time > 0 ) {
        dt = DPD_time;
        cout << "DPD Forces time: " << dt / 60 << "m" << dt % 60 << "sec" << endl;
    }
    
    if ( nList_tot_time > 0 ) {
        dt = nList_tot_time;
        cout << "NList time: " << dt / 60 << "m" << dt % 60 << "sec" << endl;
    }

	// hipStreamDestroy(stream1);
    MPI_Finalize();
	return 0;
}


int print_timestep() {
	//if (!equilData)
		//return
	int die_flag = 0;
	cout << "Step " << step << " of " << max_steps << " ";
	cout << "Global step " << global_step;


	cout << " U/V: " << Upe / V << \
		" Ubond: " << Ubond ;
	if ( n_total_angles > 0 )
	    cout << " Uangle: " << Uangle ;
	cout << " Pdiags: " << Ptens[0] << " " << Ptens[1] << " ";

	if (Dim == 3)
		cout << Ptens[2] << " ";

	dout << step << " " <<  global_step << " " << Upe << " " << Ubond << " ";
	if ( n_total_angles > 0 ) 
	    dout << Uangle << " " ;
	for (int i = 0; i < n_P_comps; i++)
		dout << Ptens[i] << " ";

	
	for (auto& Iter: Potentials)
	{
		Iter->ReportEnergies(die_flag);
	}
	cout << " UDBond: " << Udynamicbond;
	dout << " " << Udynamicbond;

	// cout << " T: " << T;
	// dout << " " << T;

    dout << endl;
	cout<<endl;
	return die_flag;

}


void run_computes(){
  compute_t_in = int(time(0));
  for (auto Iter : Computes) {
    if (step > Iter->compute_wait && step % Iter->compute_freq == 0) {
      Iter->doCompute();
    }
    check_cudaError("Compute");
  }
  compute_t_out = int(time(0));
  compute_tot_time += compute_t_out - compute_t_in;
}

void run_frame_printing() {
  // I/O blocks //
  if (traj_freq > 0 && step % traj_freq == 0) {
    print_t_in = int(time(0));
    //hipDeviceSynchronize();

    cuda_collect_x();
    write_lammps_traj();
    print_t_out = int(time(0));
    print_tot_time += print_t_out - print_t_in;
  }

  if (gsd_freq > 0 && step % gsd_freq == 0) {
    print_t_in = int(time(0));
    //hipDeviceSynchronize();

    cuda_collect_x();
    write_gsd_traj();
    print_t_out = int(time(0));
    print_tot_time += print_t_out - print_t_in;
  }

  if (grid_freq > 0 && step % grid_freq == 0) {
    print_t_in = int(time(0));
    //hipDeviceSynchronize();

    cuda_collect_rho();
    for (int i = 0; i < ntypes; i++) {
      char nm[30];
      sprintf(nm, "rho%d.dat", i);
      write_grid_data(nm, Components[i].rho);
    }

    print_t_out = int(time(0));
    print_tot_time += print_t_out - print_t_in;
  }

  if (bin_freq > 0 && step % bin_freq == 0) {
    print_t_in = int(time(0));
    //hipDeviceSynchronize();

    cuda_collect_rho();
    cuda_collect_x();

    write_binary();
    print_t_out = int(time(0));
    print_tot_time += print_t_out - print_t_in;

  }
}

void write_data_header(std::string lbl){
    dout.open(lbl);
	dout << "# step global_step Upe Ubond ";
	if ( n_total_angles > 0 )
	    dout << "Angles " ;
	if (Dim == 2)
		dout << "Pxx Pyy Pxy";
	else if (Dim == 3)
		dout << " Pxx Pyy Pzz Pxy Pxz Pyz";
	
	for (auto Iter: Potentials){
		dout << " " + Iter->potential_type;
		if (Iter->potential_type != "Charges")
			dout << Iter->type_specific_id ;
		if (Iter->potential_type == "MaierSaupe")
			dout << " Lambda" + Iter->potential_type << Iter->type_specific_id;
	}
	dout << " UDBond";
	dout << " T";

	dout << endl;

}

void set_write_status(){
	cout << "Setting frequencies" << endl;
	if (equil ){
		cout << "Setting frequencies to equilibration values" << endl;
		cout << "Equil bin freq: " << equil_bin_freq << endl;
		cout << "Equil traj freq: " << equil_traj_freq << endl;
		cout << "Equil grid freq: " << equil_grid_freq << endl;
		cout << "Equil log freq: " << equil_log_freq << endl;
		if (equilData)
			write_data_header("equil_data.dat");
		prod_bin_freq = bin_freq;
		prod_traj_freq = traj_freq;
		prod_grid_freq = grid_freq;
		prod_log_freq = log_freq;
		prod_struc_freq = struc_freq;
		
		if (equil_bin_freq > 0){
			bin_freq = equil_bin_freq;
			cout << "Equil Binary output frequency: " << equil_bin_freq << endl;
			cout << "Binary output frequency: " << bin_freq << endl;
		}
		if (equil_traj_freq > 0)
			traj_freq = equil_traj_freq;
		if (equil_grid_freq > 0){
			grid_freq = equil_grid_freq;
			cout << "Equil Grid output frequency: " << equil_grid_freq << endl;
			cout << "Grid output frequency: " << grid_freq << endl;
		}
		if (equil_log_freq > 0)	{
			cout << "Equil Log output frequency: " << equil_log_freq << endl;
			log_freq = equil_log_freq;
			cout << "Log output frequency: " << log_freq << endl;
		}
		if (equil_struc_freq > 0)
			struc_freq = equil_struc_freq;
	}
	else{
		write_data_header("data" + srank + ".dat");
		if (prod_traj_freq > 0)
			traj_freq = prod_traj_freq;
		if (prod_grid_freq > 0)
			grid_freq = prod_grid_freq;
		if (prod_bin_freq > 0)
			bin_freq = prod_bin_freq;
		if (prod_log_freq > 0)	
			log_freq = prod_log_freq;
		if (prod_struc_freq > 0)	
			struc_freq = prod_struc_freq;
	}
	if (bin_freq != 0)
		init_binary_output();
}


void set_ft_config(){
	step = 0;
	mem_use = 0;
	extra_ns_memory = 0;
	particle_sim = 0;
	field_sim = 0;

	// Global unit complex value
	I = complex<float>(0.0f, 1.0f);
	
	Dim = 2;
	delt = 0.005f;
	pmeorder = 1;
	for (int j = 0; j < Dim; j++) {
		Nx[j] = 35;
	}

	threads = 512;
	noise_mag = sqrtf(2.0f * delt);
	MAX_BONDS = 3;
	MAX_ANGLES = 3;

	max_steps = 100000;
	RAND_SEED = int(time(0));
	log_freq = 2500;
	grid_freq = 0;
	traj_freq = 0;
	gsd_freq = 0;
	struc_freq = 0;
	skip_steps = 20000;
	bin_freq = 5000;
	dump_name = "traj.lammpstrj";
	equil_name = "equil.lammpstrj";
	gsd_name = "traj.gsd";
	equil = false;
	equilData = 1;
	equil_steps = 0;
	prod_steps = 0;
	MAX_DISP = 0.0f;
	prod_bin_freq = 0;
	prod_log_freq = 0;
	prod_traj_freq = 0;
	prod_struc_freq = 0;
	prod_grid_freq = 0;
	global_step = 0;
	LOW_DENS_FLAG = 0;

}
