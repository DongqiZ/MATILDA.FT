#include "hip/hip_runtime.h"
// Copyright (c) 2023 University of Pennsylvania
// Part of MATILDA.FT, released under the GNU Public License version 2 (GPLv2).


#include "globals.h"
#include "potential_gaussian.h"
#include "device_utils.cuh"
#include <iostream>
#include <fstream>
using namespace std;

__global__ void init_device_gaussian(hipfftComplex*, hipfftComplex*, 
    float, float, const float*,
    const int, const int*, const int);
__global__ void d_multiply_cufftCpx_scalar(hipfftComplex*, float, int);
__global__ void d_complex2real(hipfftComplex*, float*, int);
__global__ void d_extractForceComp(hipfftComplex*, hipfftComplex*,
    const int, const int, const int);
__global__ void d_insertForceCompC2R(float*, hipfftComplex*, const int,
    const int, const int);

void Gaussian::Initialize() {
    Initialize_Potential();

    printf("Setting up Gaussian pair style..."); fflush(stdout);
 
    init_device_gaussian<<<M_Grid, M_Block>>>(this->d_u_k, this->d_f_k,
        initial_prefactor, sigma_squared, d_L, M, d_Nx, Dim);

    init_device_gaussian<<<M_Grid, M_Block>>>(
        this->d_master_u_k, this->d_master_f_k,
        1.0f, sigma_squared, d_L, M, d_Nx, Dim);

    hipfftExecC2C(fftplan, this->d_u_k, d_cpx1, HIPFFT_BACKWARD);
    d_complex2real<<<M_Grid, M_Block>>>(d_cpx1, this->d_u, M);

    for (int j = 0; j < Dim; j++) {
        d_extractForceComp<<<M_Grid, M_Block>>>(d_cpx1, this->d_f_k, j, Dim, M);
        hipfftExecC2C(fftplan, d_cpx1, d_cpx1, HIPFFT_BACKWARD);
        d_insertForceCompC2R<<<M_Grid, M_Block>>>(this->d_f, d_cpx1, j, Dim, M);
    }

    float k2, kv[3];

    // Define the potential and the force in k-space
    for (int i = 0; i < M; i++) {
        k2 = get_k(i, kv, Dim);

        this->u_k[i] = initial_prefactor * exp(-k2 * sigma_squared / 2.0f);
        

        for (int j = 0; j < Dim; j++) {
            this->f_k[j][i] = -I * kv[j] * this->u_k[i];
        }
            
    }
    
  
    
    InitializeVirial();
    
    printf("done!\n"); fflush(stdout);
}


Gaussian::Gaussian() {

}

Gaussian::Gaussian(istringstream &iss) : Potential(iss) {
	potential_type = "Gaussian";
	type_specific_id = num++;
    SAME_TYPE = 0;

	readRequiredParameter(iss, type1);
	readRequiredParameter(iss, type2);
	readRequiredParameter(iss, initial_prefactor);
	readRequiredParameter(iss, sigma_squared);
    if (type1 == type2){
        SAME_TYPE = 1;
    }

	// iss >> type1 >> type2 >> initial_prefactor >> sigma_squared;
    // if (iss.fail()) std::cout << "here" << std::endl;

	type1 -= 1;
	type2 -= 1;

	final_prefactor = initial_prefactor;

	sigma_squared *= sigma_squared;

	check_types();

	ramp_check_input(iss);

}

Gaussian::~Gaussian() {

}


__global__ void init_device_gaussian(hipfftComplex* uk, hipfftComplex* fk,
    float Ao, float sigma2,
    const float* dL, const int M, const int* Nx, const int Dim) {

    const int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= M)
        return;

    float k2, kv[3];

    k2 = d_get_k(ind, kv, dL, Nx, Dim);
    uk[ind].x = Ao * exp(-k2 * sigma2 / 2.0f);
    uk[ind].y = 0.f;

    for (int j = 0; j < Dim; j++) {
        fk[ind * Dim + j].x = 0.f;
        fk[ind * Dim + j].y = -kv[j] * uk[ind].x;
    }
}

void Gaussian::ReportEnergies(int& die_flag){
    static int counter = 0;
    static string reported_energy = "";
    reported_energy += " " + to_string(energy) ;
	if (std::isnan(energy)) die_flag = 1 ;
    
    if (++counter == num){
        dout << reported_energy;
        cout << " Ugauss: " + reported_energy;
        counter=0;
		reported_energy.clear();
    }
}

int Gaussian::num = 0;
