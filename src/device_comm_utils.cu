#include "hip/hip_runtime.h"
// Copyright (c) 2023 University of Pennsylvania
// Part of MATILDA.FT, released under the GNU Public License version 2 (GPLv2).


#include "globals.h"
__global__ void d_copyPositions(float*, float*, int, int);

void update_device_positions(float** out, float* d_target) {
    int i, j;
	for (i = 0; i < ns; i++) {
		for (j = 0; j < Dim; j++) {
			h_ns_float[i * Dim + j] = out[i][j];
		}
	}
	hipMemcpy(d_target, h_ns_float, ns * Dim * sizeof(float),
		hipMemcpyHostToDevice);
}

void send_3n_to_device(float** out, float *d_target) {
	int i, j;
	for (i = 0; i < ns; i++) {
		for (j = 0; j < Dim; j++) {
			h_ns_float[i * Dim + j] = out[i][j];
		}
	}
	hipMemcpy(d_target, h_ns_float, ns * Dim * sizeof(float),
		hipMemcpyHostToDevice);

	if ( Integrator::using_GJF )
		d_copyPositions<<<ns_Grid, ns_Block>>>(d_xo, d_x, Dim, ns);
}


void cuda_collect_x() {
	hipMemcpy(h_ns_float, d_x, ns * Dim * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < ns; i++)
		for (int j = 0; j < Dim; j++)
			x[i][j] = h_ns_float[i * Dim + j];

	if ( Charges::do_charges == 1 )
		hipMemcpy(charges, d_charges, ns * sizeof(float), hipMemcpyDeviceToHost);
}

void cuda_collect_rho() {
	hipMemcpy(all_rho, d_all_rho, ntypes * M * sizeof(float), 
		hipMemcpyDeviceToHost);
	// Copys all_rho to device_all_rho

	for (int i = 0; i < ntypes; i++) {
		for (int j = 0; j < M; j++) {
			Components[i].rho[j] = all_rho[i * M + j];
		}
	}
}



void cuda_collect_charge_density_field() {

	hipMemcpy(charge_density_field, d_charge_density_field, M * sizeof(float),
		hipMemcpyDeviceToHost);

	check_cudaError("cuda_collect_charge_density_field");
}

void cuda_collect_electric_field() {

	hipMemcpy(electric_field, d_electric_field, M * Dim * sizeof(float),
		hipMemcpyDeviceToHost);

	check_cudaError("cuda_collect_electric_field");
}


void cuda_collect_electrostatic_potential() {

	hipMemcpy(electrostatic_potential, d_electrostatic_potential, M * sizeof(float),
		hipMemcpyDeviceToHost);

	check_cudaError("cuda_collect_electrostatic_potential");
}


void cuda_collect_f() {
	hipMemcpy(h_ns_float, d_f, ns * Dim * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < ns; i++)
		for (int j = 0; j < Dim; j++)
			f[i][j] = h_ns_float[i * Dim + j];
}


void send_box_params_to_device() {

	// Box geometry //
	hipMemcpy(d_L, L, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Lh, Lh, 3 * sizeof(float), hipMemcpyHostToDevice);

    int nsSend = ns + extra_ns_memory;
	hipMemcpy(d_typ, tp, nsSend * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_molecID, molecID, nsSend * sizeof(int), hipMemcpyHostToDevice);

	// Bonding information //
	hipMemcpy(d_n_bonds, n_bonds, nsSend * sizeof(int), hipMemcpyHostToDevice);
	
	hipMemcpy(d_bond_req, bond_req, nbond_types * sizeof(float),
		hipMemcpyHostToDevice);
	hipMemcpy(d_bond_k, bond_k, nbond_types * sizeof(float),
		hipMemcpyHostToDevice);

	

	// Stack 2D array into 1D for transfer to device
	int* h_bond_stuff;
	h_bond_stuff = (int*)calloc(nsSend * MAX_BONDS, sizeof(int));
	int i, j;
	for (i = 0; i < nsSend; i++)
		for (j = 0; j < MAX_BONDS; j++)
			h_bond_stuff[i * MAX_BONDS + j] = bonded_to[i][j];

	hipMemcpy(d_bonded_to, h_bond_stuff, nsSend * MAX_BONDS * sizeof(int),
		hipMemcpyHostToDevice);

	for (i = 0; i < nsSend; i++)
		for (j = 0; j < MAX_BONDS; j++)
			h_bond_stuff[i * MAX_BONDS + j] = bond_type[i][j];

	hipMemcpy(d_bond_type, h_bond_stuff, nsSend * MAX_BONDS * sizeof(int),
		hipMemcpyHostToDevice);

	for (i = 0; i < nsSend; i++)
		for (j = 0; j < MAX_BONDS; j++)
			h_bond_stuff[i * MAX_BONDS + j] = bond_style[bond_type[i][j]];

	hipMemcpy(d_bond_style, h_bond_stuff, nsSend * MAX_BONDS * sizeof(int),
		hipMemcpyHostToDevice);

	free(h_bond_stuff);


    
    
    // Send angle info to device if needed
    if ( n_total_angles > 0 ) {
        hipMemcpy(d_angle_k, angle_k, nangle_types * sizeof(float), 
            hipMemcpyHostToDevice);
        hipMemcpy(d_angle_theta_eq, angle_theta_eq, nangle_types * sizeof(float), 
            hipMemcpyHostToDevice);
        hipMemcpy(d_angleIntStyle, angleIntStyle, nangle_types * sizeof(int), 
            hipMemcpyHostToDevice);

        hipMemcpy(d_n_angles, n_angles, nsSend * sizeof(int),
            hipMemcpyHostToDevice);
            

        int* h_ang;
        h_ang = (int*)calloc(nsSend * MAX_ANGLES, sizeof(int));

        for ( int i=0 ; i<nsSend ; i++ )
            for ( int j=0 ; j<MAX_ANGLES; j++ )
                h_ang[i * MAX_ANGLES + j] = angle_type[i][j];
        hipMemcpy(d_angle_type, h_ang, nsSend*MAX_ANGLES * sizeof(int),
            hipMemcpyHostToDevice);
       
        for ( int i=0 ; i<nsSend ; i++ )
            for ( int j=0 ; j<MAX_ANGLES; j++ )
                h_ang[i * MAX_ANGLES + j] = angle_first[i][j];
        hipMemcpy(d_angle_first, h_ang, nsSend*MAX_ANGLES * sizeof(int),
            hipMemcpyHostToDevice);

        for ( int i=0 ; i<nsSend ; i++ )
            for ( int j=0 ; j<MAX_ANGLES; j++ )
                h_ang[i * MAX_ANGLES + j] = angle_mid[i][j];
        hipMemcpy(d_angle_mid, h_ang, nsSend * MAX_ANGLES * sizeof(int),
            hipMemcpyHostToDevice);

        for ( int i=0 ; i<nsSend ; i++ )
            for ( int j=0 ; j<MAX_ANGLES; j++ )
                h_ang[i * MAX_ANGLES + j] = angle_end[i][j];
        hipMemcpy(d_angle_end, h_ang, nsSend*MAX_ANGLES * sizeof(int),
            hipMemcpyHostToDevice);     
             
    }// if (n_total_angles > 0)



	// Copy grid information
	hipMemcpy(d_dx, dx, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Nx, Nx, 3 * sizeof(int), hipMemcpyHostToDevice);

	// Copy masses, diffusivities
	hipMemcpy(d_mass, mass, ntypes * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Diff, Diff, ntypes * sizeof(float), hipMemcpyHostToDevice);

	// Copy charges
	if (Charges::do_charges == 1) 
		hipMemcpy(d_charges, charges, ns * sizeof(float), hipMemcpyHostToDevice);
}
